
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>

int main() {

    const int nElements = 4*1024*1024;
    size_t nbytes = nElements*sizeof(float);
    float *a_pageable, *b_pageable;
    float *a_pinned, *b_pinned;
    float *a_d;
    int ierr_a, ierr_b;
    hipDeviceProp_t prop;
    hipEvent_t startEvent, stopEvent;
    float time = 0.0;

    // pageable host memory
    a_pageable = (float*)malloc(nbytes);
    b_pageable = (float*)malloc(nbytes);

    // pinned host memory
    ierr_a = hipHostMalloc((void**)&a_pinned, nbytes, hipHostMallocDefault);
    ierr_b = hipHostMalloc((void**)&b_pinned, nbytes, hipHostMallocDefault);
    if (ierr_a != 0 || ierr_b != 0) {
        printf("Allocation of a_pinned/b_pinned failed\n");
        std::exit(1);
    }

    // initializing
    for (int i = 0; i < nElements; ++i) a_pageable[i] = i;
    memcpy(a_pinned, a_pageable, nbytes);
    memset(b_pageable, 0.0, nbytes);
    memset(b_pinned, 0.0, nbytes);

    // device memory
    hipMalloc((void**)&a_d, nbytes);

    // output device info and transfer size
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice: %s\n", prop.name);
    printf("Transfer size (MB): %f\n", nbytes/1024./1024.);

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // pageable data transfers
    printf("\nPageable transfers\n");
    hipEventRecord(startEvent, 0);
    hipMemcpy(a_d, a_pageable, nbytes, hipMemcpyHostToDevice);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("  Host to Device bandwidth (GB/s): %f\n", nbytes/time/1.e+6);

    hipEventRecord(startEvent, 0);
    hipMemcpy(b_pageable, a_d, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("  Device to Host bandwidth (GB/s): %f\n", nbytes/time/1.e+6);

    for (int i = 0; i < nElements; ++i) {
        if (a_pageable[i] != b_pageable[i]) {
            printf("*** Pageable transfers failed ***\n");
            break;
        }
    }

    hipMemset(a_d, 0.0, nbytes);

    // pinned data transfers
    printf("\nPinned transfers\n");
    hipEventRecord(startEvent, 0);
    hipMemcpy(a_d, a_pinned, nbytes, hipMemcpyHostToDevice);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("  Host to Device bandwidth (GB/s): %f\n", nbytes/time/1.e+6);

    hipEventRecord(startEvent, 0);
    hipMemcpy(b_pinned, a_d, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("  Device to Host bandwidth (GB/s): %f\n", nbytes/time/1.e+6);

    for (int i = 0; i < nElements; ++i) {
        if (a_pinned[i] != a_pinned[i]) {
            printf("*** Pinned transfers failed ***\n");
            break;
        }
    }

    printf("\n");

    // cleanup
    hipFree(a_d);
    hipHostFree(a_pinned);
    hipHostFree(b_pinned);
    free(a_pageable);
    free(b_pageable);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

}